#include "hip/hip_runtime.h"
#include "GraphPath.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include<vector>
#include<algorithm>
#include <utility>
#include <time.h>
#include<math.h>
#include"service.h"
#include"dijkstra.h"
#include"BFS.h"
#include"taskPath.h"
#include"const.h"
#include"routemask.h"
#include"PathArrange.h"
#include<fstream>
using namespace std;
#define threadsize 256

bool UDgreater(pair<int, float> elem1, pair<int, float> elem2)
{
	return elem1.second > elem2.second;
}
bool UPGservice(service s1, service s2)
{
	return s1.d>s2.d;
}
bool cmp(float a, float b)
{
	return a<b;
}
__global__ void bellmanHigh(Edge *edge, int *m, float *c, int*p, float*lambda, int*mask, int stillS)
{
	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*Task + i;
	float val = c[tail*Task + i]+1 +lambda[tid];
	if (c[biao] >val){
		*m = 1;
		c[biao] = val;
	}
}
__global__ void color(Edge *edge, int *m, float *c, int*p, float*lambda, int *mask, int stillS){

	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*Task + i;
	float val = c[tail*Task + i]+1+lambda[tid];// * pd[i];
	if (c[biao] == val){
		p[biao] = tid;
	}
}
__global__ void ChangePameterC(int*p, float*d, int* st, int taskSize, int n){
	int tid = blockIdx.y;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i >= taskSize || tid >= n)return;
	int biao = tid*taskSize + i;
	d[biao] = (st[i] == tid) ? 0.0 : 10000000000.0;
	p[biao] = -1;


}
void GraphPath::Copy2GPU(std::vector<service> &s){
	for (int i = 0; i < Task; i++)
	{
		st[i] = s[i].s;
		te[i] = s[i].t;
		pd[i] = (float)s[i].d;
	}
	for (int i = 0; i < Task; i++)
		mask[i] = i;
	for (int i = 0; i < EDge; i++)
		lambda[i] = 0;
	hipMemcpy(dev_st, st, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_te, te, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_lambda, lambda, EDge*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mask, mask, Task*sizeof(int), hipMemcpyHostToDevice);
}


GraphPath::GraphPath(Graph&_G):G(_G),StoreRoute(Task, vector<int>(1,-1)), BestRoute(Task, vector<int>())
{
	hipMalloc(&dev_edge, sizeof(Edge)*EDge);
	hipMemcpy(dev_edge, G.incL, EDge* sizeof(Edge), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_st, Task*sizeof(int));
	hipMalloc((void**)&dev_te, Task*sizeof(int));
	hipMalloc((void**)&dev_pd, Task*sizeof(float));
	hipMalloc((void**)&dev_lambda, EDge*sizeof(float));
	hipMalloc((void**)&dev_mask, Task*sizeof(int));
	hipMalloc((void**)&dev_d, Task*NODE* sizeof(float));
	hipMalloc((void**)&dev_p, Task*NODE* sizeof(int));
	hipMalloc(&dev_m, sizeof(int));
	st = new int[Task*sizeof(int)];
	te = new int[Task*sizeof(int)];
	pd = new float[Task*sizeof(float)];
	d = (float*)malloc(Task*NODE*sizeof(float));
	pre = (int*)malloc(Task*NODE*sizeof(int));
	lambda = new float[EDge*sizeof(float)];
	mask = new int[Task];
	mark = new int(1);
	capacity = (float*)malloc(EDge*sizeof(float));
	for (int i = 0; i < NODE; i++)
		{
			for (int j = 0; j < Task; j++)
			{
				if (st[j] == i)
				{
					d[i*Task + j] = 0.0;
					pre[i*Task + j] = -1;
				}
				else
				{
					d[i*Task + j] = 100000.0;
					pre[i*Task + j] = -1;
				}
			}
		}
	hipMemcpy(dev_d, d, Task*NODE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, pre, Task*NODE*sizeof(int), hipMemcpyHostToDevice);
}
vector<pair<string,float> > GraphPath::bellmanFordCuda(vector<service>&ser,ostream& Out) {
	printf("Lagrange parrel searching..............\n");
	srand(time(NULL));
	float start = float(1000*clock())/ CLOCKS_PER_SEC;
	Copy2GPU(ser);
	int num = Task;
	int mum = EDge;
	int stillS = num;
	int reme = 0;
	int count = 0;
	vector<RouteMark> bestroutes;
	devicesize += 2 * Task*sizeof(RouteMark);
	int bestround = 0;
	int zeor = 0;
	double totalflow = 0;
	for (int i = 0; i < Task; i++)
		totalflow += INFHOPS *pd[i];
	double bestadd = totalflow;
	float best = totalflow;
	vector<float>middata;
	for (int i = 0; i <1000000; i++)
	{
		count++;
		reme++;
		dim3 blocksq(Task / threadsize + 1, NODE*Task / Task);
		ChangePameterC << <blocksq, threadsize >> >(dev_p, dev_d, dev_st, Task, NODE);
		hipMemcpy(dev_lambda, lambda, EDge*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_mask, mask, Task*sizeof(int), hipMemcpyHostToDevice);
		dim3 blocks_square(stillS / threadsize + 1, EDge*Task / Task);
		do{
			hipMemcpy(dev_m, &zeor, sizeof(int), hipMemcpyHostToDevice);
			bellmanHigh << <blocks_square, threadsize >> >(dev_edge, dev_m, dev_d, dev_p, dev_lambda, dev_mask, stillS);
			hipMemcpy(mark, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		} while (*mark);
		color << <blocks_square, threadsize >> >(dev_edge, dev_m, dev_d, dev_p, dev_lambda, dev_mask, stillS);
		hipMemcpy(pre, dev_p, sizeof(int)*num*NODE, hipMemcpyDeviceToHost);
		hipMemcpy(d, dev_d, sizeof(float)*num*NODE, hipMemcpyDeviceToHost);
		int value = rearrange(&G, capacity, lambda, pre, d, pd, te, st, num, mum, bestadd, stillS, num, 1, StoreRoute, BestRoute, mask, Out, bestroutes, totalflow);
		middata.push_back(value);
		if (value<best)
		{
			bestround = count;
			best = value;
			reme = 0;
		}
		if (stillS == 0 || reme>loomore)
			break;
	}
	float end=float(1000*clock())/ CLOCKS_PER_SEC;


	vector<pair<int, vector<int>>> result = GrabResult(BestRoute, num, mum, pd);
	int addin = result.size();
	pair<float,int> tf=CheckR(&G, result,ser,string("Lag_Parallel"));
	writejsoniter(LAGPFILE,middata,string("Lag_Parallel"));
	vector<pair<string,float>> rdata;
	rdata.push_back(make_pair(string("object"),best));
	rdata.push_back(make_pair(string("inf_obj"),totalflow));
	rdata.push_back(make_pair(string("task_add_in"),addin));
	rdata.push_back(make_pair(string("flow_add_in"),tf.first));
	rdata.push_back(make_pair(string("total_weight"),tf.second));
	rdata.push_back(make_pair(string("time"),(end-start)));
	rdata.push_back(make_pair(string("iter_num"),count));
	rdata.push_back(make_pair(string("iter_time"),float(end-start)/(float)count));
	writejsondata(DATAFILE,rdata,string("Lag_Parallel"));
	return rdata;
}
void GraphPath::CudaFree(){
	hipFree(dev_st);
	hipFree(dev_te);
	hipFree(dev_pd);
	hipFree(dev_lambda);
	hipFree(dev_mask);
	hipFree(dev_d);
	hipFree(dev_p);
	hipFree(dev_m);

}
GraphPath::~GraphPath()
{
	CudaFree();
	delete[] st;
	delete[] te;
	delete[] pd;
	free(d);
	free(pre);
	delete[] lambda;
	delete[] mask;
	delete mark;
	free(capacity);
}



